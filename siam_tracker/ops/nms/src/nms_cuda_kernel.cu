#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include <stdbool.h>
#include <stdio.h>
#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) std::cout << "CUDA Error: " << \
        hipGetErrorString(XXX) << ", at line " << __LINE__ \
<< std::endl; hipDeviceSynchronize(); } while (0)

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
    float left = max(a[0], b[0]), right = min(a[2], b[2]);
    float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    float width = max(right - left, 0.f), height = max(bottom - top, 0.f);
    float interS = width * height;
    float Sa = (a[2] - a[0] ) * (a[3] - a[1]);
    float Sb = (b[2] - b[0] ) * (b[3] - b[1]);
    return interS / (Sa + Sb - interS);
}


__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


at::Tensor nms_cuda_forward(
    at::Tensor dets,
    const float nms_overlap_thresh)
{
    const int boxes_num = dets.size(0);
    const int boxes_dim = dets.size(1);

    const int col_blocks = DIVUP(boxes_num, threadsPerBlock);
    unsigned long long* mask_dev = NULL;

    CUDA_CHECK(hipMalloc(&mask_dev,
        boxes_num * col_blocks * sizeof(unsigned long long)));

    dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
        DIVUP(boxes_num, threadsPerBlock));
    dim3 threads(threadsPerBlock);

    nms_kernel<<<blocks, threads>>>(boxes_num,
        nms_overlap_thresh,
        dets.data<float>(),
        mask_dev);

    std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
    CUDA_CHECK(hipMemcpy(&mask_host[0],
                          mask_dev,
                          sizeof(unsigned long long) * boxes_num * col_blocks,
                          hipMemcpyDeviceToHost));

    std::vector<unsigned long long> remv(col_blocks);
    memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

    // we need to create a memory for keep_out on cpu
    // otherwise, the following code cannot run

    int* keep_out_cpu = new int[boxes_num];

    int num_to_keep = 0;
    for (int i = 0; i < boxes_num; i++) {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;

        if (!(remv[nblock] & (1ULL << inblock))) {
            // orignal: keep_out[num_to_keep++] = i;
            keep_out_cpu[num_to_keep++] = static_cast<float>(i);
            unsigned long long *p = &mask_host[0] + i * col_blocks;
            for (int j = nblock; j < col_blocks; j++) {
                remv[j] |= p[j];
            }
        }
    }

    auto keep_out = at::zeros({num_to_keep}, at::TensorOptions(dets.type()).dtype(at::kInt));
    // copy keep_out_cpu to keep_out on gpu
    CUDA_WARN(hipMemcpy(keep_out.data<int>(), keep_out_cpu, num_to_keep * sizeof(int), hipMemcpyHostToDevice));

    // release cuda memory
    CUDA_CHECK(hipFree(mask_dev));
    // release cpu memory
    delete []keep_out_cpu;

    return keep_out;
}
